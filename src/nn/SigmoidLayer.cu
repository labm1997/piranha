#include "hip/hip_runtime.h"

#pragma once

#include "SigmoidLayer.h"
#include "../mpc/RSS.h"
#include "../mpc/TPC.h"
#include "../mpc/FPC.h"
#include "../mpc/OPC.h"
#include "../util/Profiler.h"

#include <numeric>

extern Profiler debug_profiler;

extern nlohmann::json piranha_config;

template<typename T, template<typename, typename...> typename Share>
Profiler SigmoidLayer<T, Share>::sigmoid_profiler;

template<typename T, template<typename, typename...> typename Share>
SigmoidLayer<T, Share>::SigmoidLayer(SigmoidConfig* conf, int _layerNum, int seed) : Layer<T, Share>(_layerNum),
	conf(conf->inputDim, conf->batchSize),
	activations(conf->batchSize * conf->inputDim), 
	deltas(conf->batchSize * conf->inputDim) {

    //printf("creating sigmoid layer with input dim %d, batch size %d\n", conf->inputDim, conf->batchSize);
	activations.zero();
	deltas.zero();	
}

template<typename T, template<typename, typename...> typename Share>
void SigmoidLayer<T, Share>::loadSnapshot(std::string path) {
    // do nothing
}

template<typename T, template<typename, typename...> typename Share>
void SigmoidLayer<T, Share>::saveSnapshot(std::string path) {
    // do nothing
}

template<typename T, template<typename, typename...> typename Share>
void SigmoidLayer<T, Share>::printLayer()
{
	std::cout << "----------------------------------------------" << std::endl;
	std::cout << "(" << this->layerNum+1 << ") Sigmoid Layer\t\t  " << conf.batchSize << " x " << conf.inputDim << std::endl;
}

template<typename T, template<typename, typename...> typename Share>
void SigmoidLayer<T, Share>::forward(const Share<T> &input) {

    if (piranha_config["debug_all_forward"]) {
        printf("layer %d\n", this->layerNum);
        //printShareTensor(*const_cast<Share<T> *>(&input), "fw pass input (n=1)", 1, 1, 1, input.size() / conf.batchSize);
    }

	log_print("Sigmoid.forward");

	/*
	size_t rows = conf.batchSize; // ???
	size_t columns = conf.inputDim;
	size_t size = rows*columns;
	*/

    this->layer_profiler.start();
    sigmoid_profiler.start();
    debug_profiler.start();

    activations.zero();

    sigmoid(input, activations);

    debug_profiler.accumulate("sigmoid-fw-fprop");
    this->layer_profiler.accumulate("sigmoid-forward");
    sigmoid_profiler.accumulate("sigmoid-forward");

    if (piranha_config["debug_all_forward"]) {
        //printShareTensor(*const_cast<Share<T> *>(&activations), "fw pass activations (n=1)", 1, 1, 1, activations.size() / conf.batchSize);
        std::vector<double> vals(activations.size());
        copyToHost(activations, vals);
        
        printf("sigmoid,fw activation,min,%e,avg,%e,max,%e\n", 
                *std::min_element(vals.begin(), vals.end()),
                std::accumulate(vals.begin(), vals.end(), 0.0) / static_cast<float>(vals.size()), 
                *std::max_element(vals.begin(), vals.end()));
    }
}

template<typename T, template<typename, typename...> typename Share>
void SigmoidLayer<T, Share>::backward(const Share<T> &delta, const Share<T> &forwardInput) {

    if (piranha_config["debug_all_backward"]) {
        printf("layer %d\n", this->layerNum);
        //printShareFinite(*const_cast<Share<T> *>(&delta), "input delta for bw pass (first 10)", 10);
        std::vector<double> vals(delta.size());
        copyToHost(
            *const_cast<Share<T> *>(&delta),
            vals
        );
        
        printf("sigmoid,bw input delta,min,%e,avg,%e,max,%e\n", 
                *std::min_element(vals.begin(), vals.end()),
                std::accumulate(vals.begin(), vals.end(), 0.0) / static_cast<float>(vals.size()), 
                *std::max_element(vals.begin(), vals.end()));
    }

	log_print("Sigmoid.backward");

	sigmoid_profiler.start();
	this->layer_profiler.start();
    debug_profiler.start();

    this->deltas.zero();

	// (1) Compute backwards gradient for previous layer
	// Share<T> zeros(delta.size());
	// zeros.zero();
    // selectShare(zeros, delta, reluPrime, deltas);
    dSigmoid(activations, deltas);
    deltas *= delta;

    // (2) Compute gradients w.r.t. layer params and update
    // nothing for ReLU

    debug_profiler.accumulate("sigmoid-bw");
    sigmoid_profiler.accumulate("sigmoid-backward");
    this->layer_profiler.accumulate("sigmoid-backward");

    //return deltas;
}

template class SigmoidLayer<uint32_t, RSS>;
template class SigmoidLayer<uint64_t, RSS>;

template class SigmoidLayer<uint32_t, TPC>;
template class SigmoidLayer<uint64_t, TPC>;

template class SigmoidLayer<uint32_t, FPC>;
template class SigmoidLayer<uint64_t, FPC>;

template class SigmoidLayer<uint32_t, OPC>;
template class SigmoidLayer<uint64_t, OPC>;
